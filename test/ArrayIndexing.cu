#include "hip/hip_runtime.h"
#include <catch2/catch_test_macros.hpp>
#include "device_testmacros.cuh"
#include <cuvoxmap/utils/ArrayIndexing.hpp>

TEST_CASE("ArrayIndexing cpu", "utils")
{
    cuvoxmap::Indexing<3> indexing(2, 3, 4);

    SECTION("indexing")
    {
        REQUIRE(indexing.DIM() == 3);
        REQUIRE(indexing.getIdxSize(0) == 2);
        REQUIRE(indexing.getIdxSize(1) == 3);
        REQUIRE(indexing.getIdxSize(2) == 4);
        REQUIRE(indexing.merge({1, 2, 3}) == 23);
        REQUIRE(indexing.split(23) == std::array<uint32_t, 3UL>{1, 2, 3});

        REQUIRE(indexing.merge_device(cuvoxmap::Idx3D{1, 2, 3}) == 23);
        REQUIRE(indexing.split_device(23) == cuvoxmap::Idx3D{1, 2, 3});
    }
}

__global__ void indexing_(cuvoxmap::Indexing<3> idxing)
{
}
TEST_CASE("indexing device")
{
    cuvoxmap::Indexing<3> indexing(2, 3, 4);

    SECTION("indexing")
    {
        indexing_<<<1, 1>>>(indexing);
    }
}